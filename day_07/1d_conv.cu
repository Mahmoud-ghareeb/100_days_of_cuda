#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oneDconv(float *a, float *b, float *k, int out_width, int kernel_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < out_width)
    {
        int tmp = 0;
        for (int i=0; i<kernel_size; i++)
        {
            tmp += (a[i+idx] * k[i]);
        }
        b[idx] = tmp;
    }
}

int main()
{
    int kernel_size = 3;
    int width = 10;
    int out_width = width - kernel_size + 1;

    float *mat, *out, *k;

    mat = (float *)malloc(width * sizeof(float));
    out = (float *)malloc(out_width * sizeof(float));
    k = (float *)malloc(kernel_size * sizeof(float));

    for (int i=0; i<kernel_size; i++)
    {
        k[i] = 1;
    }

    for (int i=0; i<width; i++)
    {
        mat[i] = i+1;
    }

    float *mat_d, *out_d, *k_d;

    hipMalloc(&mat_d, width*sizeof(float));
    hipMalloc(&out_d, out_width*sizeof(float));
    hipMalloc(&k_d, kernel_size*sizeof(float));

    hipMemcpy(mat_d, mat, width*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(k_d, k, kernel_size*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16);
    dim3 gridSize((out_width+blockSize.x-1) / blockSize.x);
    oneDconv<<<gridSize, blockSize>>>(mat_d, out_d, k_d, out_width, kernel_size);

    hipMemcpy(out, out_d, out_width*sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<out_width; i++)
    {
        printf("%f ", out[i]);
    }
    
    hipFree(out_d);
    hipFree(mat_d);
    hipFree(k_d);
    free(mat);
    free(out);
    free(k);

    return 0;
}