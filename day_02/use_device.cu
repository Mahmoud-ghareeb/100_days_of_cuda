#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int add(int a, int b)
{
    return a+b;
}

__global__ void vec_add(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < n)
    {
        c[i] = add(a[i], b[i]);
    }

}

int main()
{
    int n = 6;

    int a_h[] = {1, 2, 3, 4, 5, 6};
    int b_h[] = {2, 5, 6, 7, 8, 9};
    int c_h[n];

    int *a_d, *b_d, *c_d;
    int size = n * sizeof(int);

    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

    vec_add<<<ceil(n/1024.0), 1024>>>(a_d, b_d, c_d, n);

    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    for (int i=0; i<n; i++)
    {
        printf("%d \n", c_h[i]);
    }

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}