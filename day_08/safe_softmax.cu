#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void safe_softmax(float *a, int m, int n)
{
    extern __shared__ float shMem[];
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y;

    if (y < m)
    {
        float max_v = -INFINITY;

        for (int i=0; i<n; i++)
        {
            shMem[tid*n+i] = a[y*n+i];
        }

        for (int i=0; i<n; i++)
        {
            max_v = fmaxf(shMem[tid*n+i], max_v);
        }
        
        float sum_v = 0.0f;
        for (int i=0; i<n; i++)
        {
            float tmp = expf(shMem[tid*n+i] - max_v);
            shMem[tid*n+i] = tmp;
            sum_v += tmp;
        }

        for (int i=0; i<n; i++)
        {
            shMem[tid*n+i] /= sum_v;
        }

        for (int i=0; i<n; i++)
        {
            a[y*n+i] = shMem[tid*n+i];
        }
    }
}

int main()
{
    int m = 10;
    int n = 5;
    int size = m*n*sizeof(float);
    float *a, *o;

    a = (float *) malloc(size);
    o = (float *) malloc(size);
    
    for (int i=0; i<m; i++)
    {
        for (int j=0; j<n; j++)
        {
            a[i * n + j] = i+j;
        }
    }

    float *a_d;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&a_d, size);

    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

    hipEventRecord(start);

    dim3 blockSize(1, 10);
    dim3 gridSize((m+blockSize.y-1)/blockSize.y, (m+blockSize.x-1)/blockSize.x);
    size_t sharedMemSize = sizeof(float) * blockSize.y * n;

    safe_softmax<<<blockSize, gridSize, sharedMemSize>>>(a_d, m, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time: %.3f ms\n", ms);

    hipMemcpy(o, a_d, size, hipMemcpyDeviceToHost);

    for (int i=0; i<m; i++)
    {
        for (int j=0; j<n; j++)
        {
            printf("%f  => %f \n", a[i*n+j], o[i*n+j]);
        }
    }

    hipFree(a_d);
    free(a);
    free(o);

    return 0;
}